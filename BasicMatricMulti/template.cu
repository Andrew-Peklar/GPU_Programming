#include "hip/hip_runtime.h"

#include <wb.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define T_Width 16

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
	//@@ Insert code to implement basic matrix multiplication here
	//@@ Do not use shared memory to write this kernel

	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	if ((Row < numARows) && (Col < numBColumns)) {
		float Pvalue = 0;

		for (int k = 0; k < numAColumns; k++) {
			Pvalue += A[Row*numAColumns + k] * B[k*numBColumns + Col];
			C[Row*numBColumns + Col] = Pvalue;
		}
	}
}

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostA; // The A matrix
	float *hostB; // The B matrix
	float *hostC; // The output C matrix
	float *deviceA;
	float *deviceB;
	float *deviceC;
	int numARows;    // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows;    // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows;    // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set
					 // this)
	hostC = NULL;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
	hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
	
	//@@ Set numCRows and numCColumns
	numCRows = numARows;
	numCColumns = numBColumns;

	//@@ Allocate the hostC matrix
	wbTime_stop(Generic, "Importing data and creating memory on host");
	int Allo_C	 = numCRows * numCColumns * sizeof(float);
	hostC		 = (float*)malloc(numCRows * numCColumns * sizeof(float));

	wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
	wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

	//@@ Allocate GPU memory here
	wbTime_start(GPU, "Allocating GPU memory.");
	int Allo_A = sizeof(float) * numARows * numAColumns;
	int Allo_B = sizeof(float) * numBRows * numBColumns;

	hipMalloc((void **)&deviceA, Allo_A);
	hipMalloc((void **)&deviceB, Allo_B);
	hipMalloc((void **)&deviceC, Allo_C);
	wbTime_stop(GPU, "Allocating GPU memory.");


	//@@ Copy memory to the GPU here
	wbTime_start(GPU, "Copying input memory to the GPU.");

	hipMemcpy(deviceA, hostA, Allo_A, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, Allo_B, hipMemcpyHostToDevice);

	wbTime_stop(GPU, "Copying input memory to the GPU.");
	
	//@@ Initialize the grid and block dimensions here
	dim3 DimBlock(T_Width, T_Width, 1);  						
	dim3 DimGrid((numBColumns - 1) / T_Width + 1, (numARows - 1) / T_Width + 1, 1);     

	//@@ Launch the GPU Kernel here
	wbTime_start(Compute, "Performing CUDA computation");

	matrixMultiply<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");


	//@@ Copy the GPU memory back to the CPU here
	wbTime_start(Copy, "Copying output memory to the CPU");
	hipMemcpy(hostC, deviceC, Allo_C, hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostC, numCRows, numCColumns);

	free(hostA);
	free(hostB);
	free(hostC);

	return 0;
}
